#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>
#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth; 
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );

rtDeclareVariable(float3, posA, , );
rtDeclareVariable(float3, lookA, , );
rtDeclareVariable(float3, posB, , );
rtDeclareVariable(float3, lookB, , );

rtDeclareVariable(int,			 anaglyphic, , );

rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW

static __device__ __inline__ float3 trace(float3 pos, float3 look)
{
	float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
	float3 ray_origin = pos;
	float3 ray_direction = normalize(d.x*U + d.y*V + look);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);
	return prd.result;
}

RT_PROGRAM void pinhole_camera(){
	if (anaglyphic){
		float3 olhoEsq = trace(posB, lookB);
		float3 olhoDir = trace(posA, lookA);

		float3 cyan, red;
		cyan.x = 0; cyan.y = 1; cyan.z = 1;
		red.x = 1; red.y = 0; red.z = 0;
		olhoEsq = olhoEsq * red;
		olhoDir = olhoDir * cyan;

		float3 new_color;
		new_color.x = (olhoEsq.x + olhoDir.x);
		new_color.y = (olhoEsq.y + olhoDir.y);
		new_color.z = (olhoEsq.z + olhoDir.z);
		output_buffer[launch_index] = make_color(new_color);
	}else{
		float3 res = trace(eye, W);
		output_buffer[launch_index] = make_color(res);
	}
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}
